#include "hip/hip_runtime.h"
/*!
 * Copyright 2018 H2O.ai, Inc.
 * License   Apache License Version 2.0 (see LICENSE for details)
 */

#include <thrust/device_vector.h>
#include <thrust/random.h>

#include <cub/device/device_select.cuh>

#include <random>
#include <limits>
#include <string>

#include <stdio.h>

#include "kmeans_init.cuh"

#include "KmMatrix/KmMatrix.hpp"
#include "KmMatrix/utils.cuh"
#include "KmMatrix/GpuInfo.cuh"
#include "KmMatrix/blas.cuh"


namespace H2O4GPU {
namespace KMeans {

namespace kernel {

/*
 * Compute min value for each row.
 * @tparam T Numeric type of the data
 * @param _res The output matrix with shape m x 1
 * @param _val The input matrix with shape m x n
 */
template <typename T>
__global__ void row_min_sequential(kParam<T> _res, kParam<T> _val) {

  size_t idx = global_thread_idx();
  size_t stride = grid_stride_x () * _val.cols;

  for (size_t i = idx; i < _val.size(); i += stride) {
    T min = std::numeric_limits<T>::max();

    for (size_t j = 0; j < _val.cols; ++j) {
      T tmp = _val.ptr[i+j];
      if (tmp < min)
        min = tmp;
    }

    _res.ptr[idx] = min;
  }
}

template <typename T>
__global__ void row_argmin_sequential(kParam<T> _res, kParam<T> _val) {

  size_t idx = global_thread_idx();
  size_t stride = grid_stride_x () * _val.cols;

  for (size_t i = idx; i < _val.size(); i += stride) {
    T min = std::numeric_limits<T>::max();
    int min_idx = -1;

    for (size_t j = 0; j < _val.cols; ++j) {
      T tmp = _val.ptr[i+j];
      if (tmp < min) {
        min_idx = i;
        min = tmp;
      }
    }

    _res.ptr[idx] = min_idx;
  }
}

}  // namespace kernel


template <typename T>
struct DotOp {
  void dot(KmMatrix<T>& _res, KmMatrix<T>& _val) {
    this->dot(_res, _val, _val);
  }
  void dot(KmMatrix<T>& _res, KmMatrix<T>& _lhs,
           KmMatrix<T>& _rhs) {
    constexpr T alpha = 1.0;
    constexpr T beta = 1.0;
    hipblasHandle_t handle = GpuInfo::ins().cublas_handle();
    Blas::gemm(handle,
               HIPBLAS_OP_T, HIPBLAS_OP_N,  // FIXME
               _lhs.rows(), _rhs.cols(), _lhs.cols(),
               &alpha,
               _lhs.dev_ptr(), _lhs.cols(),
               _rhs.dev_ptr(), _rhs.cols(),
               &beta,
               _res.dev_ptr(), _res.cols());
  }
};

template <typename T>
struct VecBatchDotOp {
  void dot(KmMatrix<T>& _res, KmMatrix<T>& _val) {
    this->dot(_res, _val, _val);
  }
  void dot(KmMatrix<T>& _res, KmMatrix<T>& _lhs, KmMatrix<T>& _rhs) {
    constexpr T alpha = 1.0;
    constexpr T beta = 1.0;
    hipblasHandle_t handle = GpuInfo::ins().cublas_handle();
    Blas::gemm_strided_batched(
        handle,
        // k-means use row major, so transpose the second vector.
        HIPBLAS_OP_N, HIPBLAS_OP_T,
        1, 1, _rhs.cols(),  // m, n, k
        &alpha,
        _lhs.dev_ptr(), 1, _lhs.cols(),
        _rhs.dev_ptr(), 1, _rhs.cols(),
        &beta,
        _res.dev_ptr(), _res.cols(), 1,  // c should be columun vector
        _lhs.rows());
  }
};

// FIXME: Using struct for operations is just keeping the possibility of
// creating an unified operations for KmMatrix. For example, let KmMatrix
// inherit those left associative ops, or create an inferface for elementwise
// operations.
template <typename T>
struct SumOp {
  T sum(KmMatrix<T>& _val) {
    T* raw_ptr = _val.dev_ptr();
    thrust::device_ptr<T> ptr (raw_ptr);
    T res = thrust::reduce(ptr, ptr + _val.size(), (T)0, thrust::plus<T>());
    return res;
  }
};

template <typename T>
struct MeanOp {
  T mean(KmMatrix<T>& _val) {
    T res = SumOp<T>().sum(_val);
    return res;
  }
};

template <typename T>
struct MulOp {
  void mul(KmMatrix<T>& _res, KmMatrix<T>& _lhs, T _rhs) {
    hipblasHandle_t handle = GpuInfo::ins().cublas_handle();
    Blas::axpy(
        handle, _lhs.size(),  // handle, n
        &_rhs,                // alpha
        _lhs.dev_ptr(), 1,
        _res.dev_ptr(), 1);
  }
};

template <typename T>
struct ArgMinOp {
  void argmin(KmMatrix<T>& _res, KmMatrix<T>& _val, KmMatrixDim _dim) {
    size_t blocks = GpuInfo::ins().blocks(32);
    if (_dim == KmMatrixDim::ROW) {
      kernel::row_argmin_sequential<<<blocks, 256, sizeof(T)*_val.cols()>>>(
          _res.k_param(), _val.k_param());
    } else {
      // FIXME
      M_ERROR("Not implemented");
    }
  }
};

template <typename T>
struct MinOp {

  void min(KmMatrix<T>& _res, KmMatrix<T>& _val, KmMatrixDim _dim) {
    size_t blocks = GpuInfo::ins().blocks(32);
    if (_dim == KmMatrixDim::ROW) {
      kernel::row_min_sequential<<<blocks, 256, sizeof(T)*_val.cols()>>>(
          _res.k_param(), _val.k_param());
    } else {
      // FIXME
      M_ERROR("Not implemented");
    }
  }
};

namespace kernel {
// X^2 + Y^2, here only calculates the + operation.
template <typename T>
__global__ void construct_distance_pairs_kernel(
    kParam<T> _distance_pairs,
    kParam<T> _data_dots, kParam<T> _centroids_dots) {

  size_t idx = global_thread_idx();  // indexing data
  size_t idy = global_thread_idy();  // indexing centroids

  // FIXME: Is using shared memory necessary?

  size_t stride_x = grid_stride_x () * _data_dots.cols;
  // strides only for data.
  for (size_t i = idx; i < _data_dots.rows; i += stride_x) {
    if (i < _data_dots.rows && idy < _centroids_dots.rows ) {
      // i + idy: x^2 + y^2 between i^th data (a.k.a x) and idy^th
      // centroid (a.k.a y)
      _distance_pairs.ptr[i + idy] =
          _data_dots.ptr[idx] + _centroids_dots.ptr[idy];
    }
  }
}

}  // namespace kernel

// Extracted as an independent Op for k-means use.
template <typename T>
struct PairWiseDistanceOp {
  KmMatrix<T> data_dot_;
  KmMatrix<T> centroids_dot_;
  KmMatrix<T> distance_pairs_;

  bool initialized_;

  void initialize(size_t _n_data, size_t k, size_t _dim) {
    // FIXME
  }

  PairWiseDistanceOp () : initialized_(false) {}

  PairWiseDistanceOp (KmMatrix<T>& _data_dot, KmMatrix<T>& _centroids_dot,
                      KmMatrix<T>& _distance_pairs) :
      data_dot_(_data_dot), centroids_dot_(_centroids_dot),
      distance_pairs_(_distance_pairs), initialized_(true) {
    data_dot_.set_name ("data dot");
    centroids_dot_.set_name ("centroids_dot");
    distance_pairs_.set_name ("distance pairs");
  }

  KmMatrix<T> operator()(KmMatrix<T>& _data, KmMatrix<T>& _centroids) {

    kernel::construct_distance_pairs_kernel<<<
        dim3(GpuInfo::ins().blocks(32), div_roundup(_centroids.rows(), 16)),
        dim3(32, 16)>>>(  // FIXME: Tune this.
            distance_pairs_.k_param(),
            data_dot_.k_param(),
            centroids_dot_.k_param());

    CUDA_CHECK(hipGetLastError());

    hipblasHandle_t handle = GpuInfo::ins().cublas_handle();

    T alpha = -2.0;
    T beta = 1.0;

    Blas::gemm(
        handle,
        HIPBLAS_OP_T, HIPBLAS_OP_N,
        // n, d, d/k
        _data.rows(), _data.cols(), _data.cols(),
        &alpha,
        _data.dev_ptr(), _data.rows(),
        _centroids.dev_ptr(), _centroids.cols(),
        &beta,
        distance_pairs_.dev_ptr(), distance_pairs_.rows());

    return distance_pairs_;
  }
};


template <typename T>
KmMatrix<T> KmeansLlInit<T>::probability(
    KmMatrix<T>& _data, KmMatrix<T>& _centroids) {

  KmMatrix<T> centroids_dot (_centroids.rows(), 1);

  VecBatchDotOp<T>().dot(centroids_dot, _centroids);

  // FIXME: Time this
  distance_pairs_ = KmMatrix<T>(_data.rows(), _centroids.rows());
  PairWiseDistanceOp<T> distance_op (data_dot_, centroids_dot, distance_pairs_);
  distance_pairs_ = distance_op(_data, _centroids);

  KmMatrix<T> min_distances (_data.rows(), 1);

  MinOp<T>().min(min_distances, distance_pairs_, KmMatrixDim::ROW);

  T cost = SumOp<T>().sum(min_distances);

  KmMatrix<T> prob (min_distances.rows(), 1);
  MulOp<T>().mul(prob, min_distances, over_sample_ / cost);

  return prob;
}


template <typename T>
KmMatrix<T> KmeansLlInit<T>::sample_centroids(
    KmMatrix<T>& _data, KmMatrix<T>& _prob) {

  KmMatrix<T> thresholds = generator_->generate(_data.rows());

  T * thresholds_ptr = thresholds.dev_ptr();

  // If use kParam, nvcc complains:
  // identifier "H2O4GPU::KMeans::kParam<double> ::kParam" is undefined in
  // device code.
  T* prob_ptr = _prob.dev_ptr();

  auto prob_iter = thrust::make_counting_iterator(0);
  size_t n_new_centroids = thrust::count_if(
      thrust::device, prob_iter,
      prob_iter + _prob.size(),
      [=] __device__ (int idx) {
        float thresh = thresholds_ptr[idx];
        T prob_x = prob_ptr[idx];
        return prob_x > thresh;
      });

  KmMatrix<T> new_centroids(n_new_centroids, _data.cols());
  thrust::device_ptr<T> new_centroids_ptr (new_centroids.dev_ptr());

  thrust::device_ptr<T> data_ptr (_data.dev_ptr());

  size_t cols = _data.cols();
  // renew iterator
  prob_iter = thrust::make_counting_iterator(0);
  thrust::copy_if(
      thrust::device,
      data_ptr, data_ptr + _data.size(), prob_iter,
      new_centroids_ptr,
      [=] __device__(int idx) {
        size_t row = idx / cols;
        T thresh = thresholds_ptr[row];
        T prob_x = prob_ptr[row];
        return prob_x > thresh;
      });

  return new_centroids;
}

template <typename T>
KmMatrix<T>
KmeansLlInit<T>::operator()(KmMatrix<T>& _data, size_t _k) {

  if (seed_ < 0) {
    std::random_device rd;
    seed_ = rd();
  }
  k_ = _k;

  std::mt19937 generator(0);

  std::uniform_int_distribution<> distribution(0, _data.rows());
  size_t idx = distribution(generator);

  // Calculate X^2 (point-wise)
  data_dot_ = KmMatrix<T>(_data.rows(), 1);
  VecBatchDotOp<T>().dot(data_dot_, _data);

  // First centroid
  KmMatrix<T> centroids = _data.row(idx);

  KmMatrix<T> prob = probability(_data, centroids);

  T cost = SumOp<T>().sum(prob);

  for (size_t i = 0; i < std::log(cost); ++i) {
    prob = probability(_data, centroids);
    KmMatrix<T> new_centroids = sample_centroids(_data, prob);
    centroids = stack(centroids, new_centroids, KmMatrixDim::ROW);
  }

  if (centroids.rows() < k_) {
    // FIXME: When n_centroids < k
    // Get random selection in?
  }

  // FIXME: re-cluster
  // kmeans_plus_plus(centroids);
  return centroids;
}

#define INSTANTIATE(T)                                          \
  template KmMatrix<T> KmeansLlInit<T>::operator()(             \
      KmMatrix<T>& _data, size_t _k);                           \
  template KmMatrix<T> KmeansLlInit<T>::probability(            \
      KmMatrix<T>& data, KmMatrix<T>& centroids);               \
  template KmMatrix<T> KmeansLlInit<T>::sample_centroids(       \
      KmMatrix<T>& data, KmMatrix<T>& centroids);               \

INSTANTIATE(float)
INSTANTIATE(double)
// FIXME: int is not supported due to random kernel

}  // namespace Kmeans
}  // namespace H2O4GPU
